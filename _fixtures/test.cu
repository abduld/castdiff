
#include <hip/hip_runtime.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if( i < len ) out[i] = in1[i] + in2[i];
}