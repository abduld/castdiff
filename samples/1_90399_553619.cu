#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>

#define wbCheck(stmt)  do {                                                \
                        hipError_t err = stmt;                            \
                        if (err != hipSuccess) {                          \
                            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
                            return -1;                                     \
                        }                                                  \
                       } while(0)


__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	
	int i = threadIdx.x+blockDim.x*blockIdx.x;
    if(i<len) out[i] = in1[i] + in2[i];  
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;
	
	hipError_t err;
	int size;
	

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
	
	size = inputLength * sizeof(float);
	
	wbCheck(hipMalloc((void **)&deviceInput1, size));
	
	/*
	err = hipMalloc((void **)&deviceInput1, size);
	if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
	*/

	err = hipMalloc((void **)&deviceInput2, size);
	if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
	
	err = hipMalloc((void **)&deviceOutput, size);
	if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
	
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
	
	err = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
	
    err = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
	
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
	
	dim3 DimGrid((inputLength-1)/256 + 1, 1, 1); 
	dim3 DimBlock(256, 1, 1);
    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
	
	vecAdd<<<DimGrid,DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
	
	err = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
	
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here

	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);
	
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

