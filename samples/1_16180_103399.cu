
#include <hip/hip_runtime.h>
__global__ 
void vecAdd(float *in1, float *in2, float *out, int len) {
    
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i<len ) {
		out[i] = in1[i]+in2[i];
	}
}
