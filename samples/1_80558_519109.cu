#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>
#include	<hip/hip_runtime.h>

#define wbCheck(stmt) do {                                                \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
        wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
        wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
        return -1;                                                        \
    }                                                                     \
} while(0)

__global__ 
void vecAdd(float *in1, float *in2, float *out, int len) {
    
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i<len ) {
		out[i] = in1[i]+in2[i];
	}
}

__host__
int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
	int numBytes;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
	numBytes = (sizeof(float) * inputLength);
    hostOutput = (float *) malloc(numBytes);
    wbTime_stop(Generic, "Importing data and creating memory on host");

	wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc(&deviceInput1, numBytes));
	wbCheck(hipMalloc(&deviceInput2, numBytes));
	wbCheck(hipMalloc(&deviceOutput, numBytes));
	
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput1, hostInput1, numBytes, hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceInput2, hostInput2, numBytes, hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    dim3 DimGrid((inputLength-1)/256 + 1, 1, 1);
	dim3 DimBlock(256, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    vecAdd<<<DimGrid,DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
	wbCheck(hipGetLastError());

    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numBytes, hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    wbCheck(hipFree(deviceInput1));
	wbCheck(hipFree(deviceInput2));
	wbCheck(hipFree(deviceOutput));

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

