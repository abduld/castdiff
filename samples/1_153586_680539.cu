#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>
#include    <hip/hip_runtime.h>
#include    <stdio.h>
#include    <stdlib.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int i;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i<len)
	{
		out[i] = in1[i] + in2[i];
	}
						
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);
	hipError_t err;
	int size;

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);
	size = inputLength * sizeof(float);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
	err = hipMalloc((void**)&deviceInput1, size);
	if(err != hipSuccess)
	{
		printf("\nError...! Can't allocate memory for input array1 in device\n");
	}
	err = hipMalloc((void**)&deviceInput2, size);
	if(err != hipSuccess)
	{
		printf("\nError...! Can't allocate memory for input array2 in device\n");
	}
	err = hipMalloc((void**)&deviceOutput, size);
	if(err != hipSuccess)
	{
		printf("\nError...! Can't allocate memory for output array in device\n");
	}

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
	err = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
		printf("\nError...! Couldn't copy input array1 from host to device\n");
	
	err = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
		printf("\nError...! Couldn't copy input array2 from host to device\n");

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
	dim3 Grid(((inputLength-1)/256)+1,1,1);
	dim3 Block(256,1,1);

    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd<<<Grid, Block>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
	err = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
		printf("\nError...! Couldn't copy output array from device to host\n");
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

