#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<len)
		out[i] = in1[i]+in2[i];
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
	float* d_A;
	float* d_B;
	float* d_C;
    int size = inputLength*sizeof(float);
	hipMalloc((void**) &d_A,size);
	hipMalloc((void**) &d_B,size);
	hipMalloc((void**) &d_C,size);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
	hipMemcpy(d_A,hostInput1,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,hostInput2,size,hipMemcpyHostToDevice);


    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
	dim3 DimGrid(((inputLength-1)/256 +1.0),1,1);
	dim3 DimBlock(256,1,1);
	


    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
	vecAdd<<<DimGrid,DimBlock>>>(d_A,d_B,d_C,inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput,d_C,size,hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

