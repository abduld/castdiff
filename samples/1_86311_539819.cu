#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;
	
    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	const int memSize = inputLength * sizeof(float);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void **) &deviceInput1, memSize));
    wbCheck(hipMalloc((void **) &deviceInput2, memSize));
    wbCheck(hipMalloc((void **) &deviceOutput, memSize));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceInput1, hostInput1, memSize, hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceInput2, hostInput2, memSize, hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    const int threadsCntPerBlock = 1024;
    dim3 DimGrid((inputLength - 1) / threadsCntPerBlock + 1, 1, 1);
    dim3 DimBlock(threadsCntPerBlock, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostOutput, deviceOutput, memSize, hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceInput1));
    wbCheck(hipFree(deviceInput2));
    wbCheck(hipFree(deviceOutput));

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

