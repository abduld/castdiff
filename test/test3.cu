#include <hip/hip_runtime.h>

__device__ int outInvariant(int inValue) {
  return inValue * inValue;
}

__host__ __device__ int outDependent(int value, int inIdx, int outIdx) {
  if (inIdx == outIdx) {
    return 2 * value;
  } else if (inIdx > outIdx) {
    return value / (inIdx - outIdx);
  } else {
    return value / (outIdx - inIdx);
  }
}

