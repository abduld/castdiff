#include <hip/hip_runtime.h>

__global__ void vecadd(int * a, int * b, int len) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	a[idx] += b[idx];
}

