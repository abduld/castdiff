#include "hip/hip_runtime.h"
// MP 1
#include    <wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<len) out[idx] = in1[idx] + in2[idx];
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength, " elements");


    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    int byteSize =sizeof(float) * inputLength;

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here

    hipMalloc((void **) &deviceInput1, byteSize);
    hipMalloc((void **) &deviceInput2, byteSize);
    hipMalloc((void **) &deviceOutput, byteSize);


    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    hipMemcpy(deviceInput1, hostInput1, byteSize,hipMemcpyHostToDevice);

    hipMemcpy(deviceInput2, hostInput1, byteSize,hipMemcpyHostToDevice);


    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
     int block_size = 16;
     int n_blocks = inputLength /block_size + (inputLength%block_size == 0 ? 0:1);


    vecAdd<<< n_blocks, block_size>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);


    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, byteSize,hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here


    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
