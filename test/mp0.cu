
#include <hip/hip_runtime.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<len) out[idx] = in1[idx] + in2[idx];
}